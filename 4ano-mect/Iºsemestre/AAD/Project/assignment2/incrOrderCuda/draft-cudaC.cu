#include "hip/hip_runtime.h"
__global__ static void order_array_cuda_kernel(unsigned int *data, unsigned int length)
{
    /* compute thread number */
    unsigned int x, y, idx;

    x = (unsigned int)threadIdx.x + (unsigned int)blockDim.x * (unsigned int)blockIdx.x;
    y = (unsigned int)threadIdx.y + (unsigned int)blockDim.y * (unsigned int)blockIdx.y;
    idx = (unsigned int)blockDim.x * (unsigned int)gridDim.x * y + x;

    if (idx >= N_ARRAYS)
        return; // safety precaution

    /* adjust pointer to the array to be ordered */
    data += idx;

    for (unsigned int i = 0; i < length; i++)
    {

        unsigned int offset = i % 2;
        unsigned int leftIndex = 2 * idx + offset;
        unsigned int rightIndex = leftIndex + 1;

        if (rightIndex < length)
        {
            if (shouldSwap(data[leftIndex], data[rightIndex]))
            {
                swap(&data[leftIndex], &data[rightIndex]);
            }
        }
        __syncthreads();
    }
}

__host__ __device__ static void swap(int *a, int *b)
{
    int tmp = *a;
    *a = *b;
    *b = tmp;
}
__host__ __device__ bool ShouldSwap<int>::operator()(const int left, const int right) const
{
    return left > right;
}

//---------------------------------------------

// CUDA kernel - even comparisons
__global__ void even_swapper(int *data, int length)
{
    unsigned int x, y, idx;
    x = (unsigned int)threadIdx.x + (unsigned int)blockDim.x * (unsigned int)blockIdx.x;
    y = (unsigned int)threadIdx.y + (unsigned int)blockDim.y * (unsigned int)blockIdx.y;
    idx = (unsigned int)blockDim.x * (unsigned int)gridDim.x * y + x;

    if (idx >= N_ARRAYS)
        return; // safety precaution

    /* adjust pointer to the array to be ordered */
    data += idx;

    if (idx % 2 == 0 && idx < length - 1)
    {
        if (data[idx + 1] < data[idx])
        {
            // switch in the data array
            int temp = data[idx];
            data[idx] = data[idx + 1];
            data[idx + 1] = temp;
        }
    }
}

// CUDA kernel - odd comparisons
__global__ void odd_swapper(int *data, int length)
{
    unsigned int x, y, idx;
    x = (unsigned int)threadIdx.x + (unsigned int)blockDim.x * (unsigned int)blockIdx.x;
    y = (unsigned int)threadIdx.y + (unsigned int)blockDim.y * (unsigned int)blockIdx.y;
    idx = (unsigned int)blockDim.x * (unsigned int)gridDim.x * y + x;

    if (idx >= N_ARRAYS)
        return; // safety precaution

    /* adjust pointer to the array to be ordered */
    data += idx;

    if (idx % 2 != 0 && idx < length - 2)
    {
        if (data[idx + 1] < data[idx])
        {
            // switch in the data array
            int temp = data[idx];
            data[idx] = data[idx + 1];
            data[idx + 1] = temp;
        }
    }
}