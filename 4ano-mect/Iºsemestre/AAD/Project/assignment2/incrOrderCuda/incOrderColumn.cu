#include "hip/hip_runtime.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>

#include "common.h"
#include <hip/hip_runtime.h>

/* program configuration */

#ifndef ARRAY_LENGTH
# define ARRAY_LENGTH  (1 << 10)
#endif
#ifndef N_ARRAYS
# define N_ARRAYS  (1 << 10)
#endif

/* allusion to internal functions */
static void order_array_cpu_kernel (unsigned int *data, unsigned int length);
__global__ static void order_array_cuda_kernel (unsigned int *data, unsigned int length);
static double get_delta_time(void);

/* main program */

int main (int argc, char **argv)
{
  printf("%s Starting...\n", argv[0]);
  if (sizeof (unsigned int) != (size_t) 4)
     { fprintf (stderr, "Type int is not 4 bytes long.");
       return EXIT_FAILURE;
     }

  /* set up device */
  int dev = 0;
  hipDeviceProp_t deviceProp;

  CHECK (hipGetDeviceProperties (&deviceProp, dev));
  printf("Using Device %d: %s\n", dev, deviceProp.name);
  CHECK (hipSetDevice (dev));

  /* allocate memory space in host and device to store the arrays */
  unsigned int data_size;
  unsigned int *host_data;
  unsigned int *device_data;

  data_size = (size_t) N_ARRAYS * (size_t) ARRAY_LENGTH * sizeof (unsigned int);
  if (data_size > (size_t) 1.8e9)
     { fprintf (stderr, "Too much memory to be allocated!\n");
       return EXIT_FAILURE;
     }
  printf ("Total data size: %u\n", data_size);

  host_data = (unsigned int *) malloc (data_size);
  CHECK (hipMalloc ((void **) &device_data, data_size));

  /* initialize host data */
  int i;

  (void) get_delta_time ();
  srand(0xAAD2022);
  for (i = 0; i < (int) (data_size / (int) sizeof(unsigned int)); i++)
    host_data[i] = (unsigned int) ((double) rand ()/RAND_MAX * 10000);
  printf ("The initialization of host data took %.3e seconds\n",get_delta_time ());

  /* copy host data to device memory */
  (void) get_delta_time ();
  CHECK (hipMemcpy (device_data, host_data, data_size, hipMemcpyHostToDevice));
  printf ("The transfer of %d bytes from the host to the device took %.3e seconds\n", data_size, get_delta_time ());

  /* run the computational kernel on the GPU
     as an example, N_ARRAYS threads are launched where each thread deals with one array
  */
  unsigned int gridDimX,gridDimY,gridDimZ,blockDimX,blockDimY,blockDimZ;

  blockDimX = 1 << 0;                                             // optimize!
  blockDimY = 1 << 0;                                             // optimize!
  blockDimZ = 1 << 0;                                             // do not change!
  gridDimX = 1 << 10;                                             // optimize!
  gridDimY = 1 << 0;                                              // optimize!
  gridDimZ = 1 << 0;                                              // do not change!

  dim3 grid (gridDimX, gridDimY, gridDimZ);
  dim3 block (blockDimX, blockDimY, blockDimZ);

  if ((gridDimX * gridDimY * blockDimX * blockDimY) != N_ARRAYS)
     { fprintf (stderr, "Wrong configuration!\n");
       return EXIT_FAILURE;
     }
  (void) get_delta_time ();
  order_array_cuda_kernel <<<grid, block>>> (device_data, ARRAY_LENGTH);
  CHECK (hipDeviceSynchronize ());                               // wait for kernel to finish
  CHECK (hipGetLastError ());                                    // check for kernel errors
  printf("The CUDA kernel <<<(%d,%d,%d), (%d,%d,%d)>>> took %.3e seconds to run\n",
         gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, get_delta_time ());

  /* copy result back to host side */
  unsigned int *modified_device_data;

  modified_device_data = (unsigned int *) malloc (data_size);
  CHECK (hipMemcpy (modified_device_data, device_data, data_size, hipMemcpyDeviceToHost));
  printf ("The transfer of %ld bytes from the device to the host took %.3e seconds\n", (long) data_size, get_delta_time ());

  /* free device allocated memory */
  CHECK (hipFree (device_data));

  /* reset device */
  CHECK (hipDeviceReset ());

  /* run the computational kernel on the CPU */
  (void) get_delta_time ();
  for (i = 0; i < N_ARRAYS; i++)
    order_array_cpu_kernel (host_data + i, ARRAY_LENGTH);
  printf("The cpu kernel took %.3e seconds to run (single core)\n", get_delta_time ());

  /* compare results from CPU and GPU */
  for(i = 0; i < N_ARRAYS * ARRAY_LENGTH; i++)
    if (host_data[i] != modified_device_data[i])
       { fprintf (stderr, "Mismatch in array %d, element %d\n", i / N_ARRAYS, i % ARRAY_LENGTH);
         return EXIT_FAILURE;
       }
  printf ("All is well!\n");

  /* free host memory */
  free (host_data);
  free (modified_device_data);

  return EXIT_SUCCESS;
}

static void order_array_cpu_kernel (unsigned int *data, unsigned int length)
{
  unsigned int i, j;
  unsigned int tmp;
  bool noSwap;

  for (i = 0; i < length - 1; i++)
  { noSwap = true;
    for (j = length - 1; j > i; j--)
      if (data[j*N_ARRAYS] < data[(j-1)*N_ARRAYS])
         { tmp = data[j*N_ARRAYS];
           data[j*N_ARRAYS] = data[(j-1)*N_ARRAYS];
           data[(j-1)*N_ARRAYS] = tmp;
           noSwap = false;
         }
    if (noSwap) break;
  }
}

__global__ static void order_array_cuda_kernel (unsigned int *data, unsigned int length)
{
  /* compute thread number */
  unsigned int x, y, idx;

  x = (unsigned int) threadIdx.x + (unsigned int) blockDim.x * (unsigned int) blockIdx.x;
  y = (unsigned int) threadIdx.y + (unsigned int) blockDim.y * (unsigned int) blockIdx.y;
  idx = (unsigned int) blockDim.x * (unsigned int) gridDim.x * y + x;
  if (idx >= N_ARRAYS)
     return;                                             // safety precaution

  /* adjust pointer to the array to be ordered */
  data += idx;

  /* order the array elements */
  unsigned int i, j;
  unsigned int tmp;
  bool noSwap;

  for (i = 0; i < length - 1; i++)
  { noSwap = true;
    for (j = length - 1; j > i; j--)
      if (data[j*N_ARRAYS] < data[(j-1)*N_ARRAYS])
         { tmp = data[j*N_ARRAYS];
           data[j*N_ARRAYS] = data[(j-1)*N_ARRAYS];
           data[(j-1)*N_ARRAYS] = tmp;
           noSwap = false;
         }
    if (noSwap) break;
  }
}

static double get_delta_time(void)
{
  static struct timespec t0,t1;

  t0 = t1;
  if(clock_gettime(CLOCK_MONOTONIC,&t1) != 0)
  {
    perror("clock_gettime");
    exit(1);
  }
  return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}
